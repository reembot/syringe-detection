#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/imgproc.hpp"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <vector>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;
using namespace std;

#include <pthread.h>
#include <sched.h>

#define NUM_THREADS 4

typedef struct {
    int threadIdx;
    Mat roisrc;
    Mat prevroi;
    int countPr;
    int countR;
    int countPh;
    int countL;
    int countW;
    int w1x;
    int w1y;
    int framenum;
}  threadParams_t;

// POSIX thread declarations and scheduling attributes
//
pthread_t threads[NUM_THREADS];
threadParams_t threadParams[NUM_THREADS];


// See www.asciitable.com
#define ESCAPE_KEY (27)
#define SYSTEM_ERROR (-1)

__global__ void mykernel(unsigned char *roirow){
    unsigned int tid = threadIdx.x;
    float bval,gval,rval;
    unsigned int i = blockIdx.x*blockDim.x+tid;
    if(i<2027520 && i%3==0) 
    {
      bval=(float)roirow[i];
      gval=(float)roirow[i+1];
      rval=(float)roirow[i+2];

//    if(abs(bval-bval2) < 30 && abs(gval-gval2) < 30 && abs(gval-gval2) < 30)
    if(1)
    {
      if ( (rval > 96) && (rval-bval > rval/3) && (bval < 32)  && (gval < rval * 9 / 10) && (gval > 3 * rval / 5) ) 
      {
        roirow[i]=1;
      }   
      else if ( (rval > 224) && (bval < 64) && (gval < 96) ) 
      {   
        roirow[i]=2; 
      } 
      else if ( (rval > 128) && (bval > 128) && (gval < 3 * rval / 4) && (gval > rval / 2) ) 
      {
        roirow[i]=3;
      } 
      else if ( (abs(rval-gval) < 20) && (abs(rval-bval) < 20) && (abs(gval-bval) < 20) ) 
      {
        if ( (rval > 48) && (gval > 48) && (bval > 48) && (rval < 80) && (gval < 80) && (bval < 80) ) 
        {
          roirow[i]=4;
        } 
        else if ( (rval > 175) && (gval > 175) && (bval > 175) ) 
        {
          roirow[i]=5;
        }
        else
        {
          roirow[i]=0;
        }
      }
      else
      {
        roirow[i]=0;
      }
    }
    else
    {
      roirow[i]=0;
    }
    }
    
}

void *findLabelThread(void *threadp)
{
    hipError_t cuda_ret;
    threadParams_t *threadParams = (threadParams_t *)threadp;
    int size = 16*sizeof(int);
    unsigned char rowvals[2027520];
//    for(int i=0;i<506880;i++) rowvals[i]=0;
    unsigned char *gpuroirow;
    cuda_ret=hipMalloc((void **)&gpuroirow,2027520*sizeof(unsigned char));
    hipMemcpy(gpuroirow,threadParams->roisrc.ptr<uchar>(0),2027520*sizeof(unsigned char),hipMemcpyHostToDevice);
//         hipDeviceSynchronize();
//    hipStreamSynchronize(0);
    mykernel<<<1980,1024>>>(gpuroirow);
//    hipDeviceSynchronize();
//    hipStreamSynchronize(0);
//         cuda_ret = hipGetLastError();
    hipMemcpy(rowvals,gpuroirow,2027520*sizeof(unsigned char),hipMemcpyDeviceToHost);
//    hipDeviceSynchronize();
//    hipStreamSynchronize(0);
    for(int i=0;i<2027520;i+=3) 
    {
      switch(rowvals[i])
      {
        case 1:
          threadParams->countPr++;
          break;
        case 2:
          threadParams->countR++;
          break;
        case 3:
          threadParams->countPh++;
          break;
        case 4:
          threadParams->countL++;
          break;
        case 5:
          threadParams->countW++;
          break;
      }
    }

    hipFree(gpuroirow);
//    hipStreamSynchronize(0);

}

int main(int argc, char** argv)
{
   const char* default_file = "0";
   const char* output_file = "vout.csv";
   const char* filename = argc >=2 ? argv[1] : default_file;
   const char* out_filename = argc >=3 ? argv[2] : output_file;
   ofstream outfile;
   outfile.open(out_filename);
   VideoCapture cap;
   int waitval = 1;
   if (*filename == '0') {
     cap.open(0);
   } else {
     cap.open(filename);
   }
   bool origView=1;
   bool hadRoc=0,hadProp=0,hadPhen=0,hadLid=0,hadOd=0,hadDex=0;
   int pW1x=0,pW1y=0,pW2x=0,pW2y=0;
   int prevStamp=0;

/*
   namedWindow(filename);
   namedWindow("zoomed");
   namedWindow("canned");
*/

   char winInput;

   Mat src, roisrc, srcSeg, srcSeg2, srcSegSum, prevroi, srcclone, dst, gray,threshed,canned,subshow;
   Mat matPr,matR,matPh,matL,matD;
   int showMat = 0;

   cap.read(src);
   if(src.empty())
   {
     return 0;
   }
   srcSeg = src.clone();
   srcSegSum = src.clone();
   dst = src.clone();
   cvtColor(dst, gray, COLOR_BGR2GRAY);
   Canny(gray, canned, 35, 70);
   int cnt=0,circnt=0;

//   VideoWriter vout;
//   Size S = Size((int) cap.get(CAP_PROP_FRAME_WIDTH),    // Acquire input size
//                  (int) cap.get(CAP_PROP_FRAME_HEIGHT));

//   vout.open("colors.mp4", VideoWriter::fourcc('H','2','6','4'), cap.get(CAP_PROP_FPS), S, true);
//   vout.open(argv[3], VideoWriter::fourcc('H','2','6','4'), 10, Size(1280,800), true);

//   printf("Press space to pause and unpause, 's' to step through each frame, 'c' to switch views or 'esc' to exit\n");

   while (1)
   {
      if(cnt>594){return(0);}
      float bval=0,gval=0,rval=0,bval2=0,gval2=0,rval2=0;
      
      int rowbuff = 75;
      int W1x=0,W1y=960,W2x=0,W2y=0;

      int countR=0,countPr=0,countPh=0,countL=0,countW=0;
      bool hasRoc=0,hasProp=0,hasPhen=0,hasLid=0,hasOd=0,hasDex=0;

      for(int i=0; i<NUM_THREADS; i++)
      {
        cnt++;
        cap.read(src);
        if(src.empty())
        {
          return(0);
        }
      
        roisrc = src(Rect(150,75,768,880)).clone();





          threadParams[i].threadIdx=i;
          threadParams[i].roisrc=roisrc;
          threadParams[i].prevroi=prevroi;
          threadParams[i].countPr=0;
          threadParams[i].countR=0;
          threadParams[i].countPh=0;
          threadParams[i].countL=0;
          threadParams[i].countW=0;
          threadParams[i].w1x=0;
          threadParams[i].w1y=0;
          threadParams[i].framenum=cnt;
          pthread_create(&threads[i],   // pointer to thread descriptor
//            (void *)0,     // use default attributes
            NULL,
            findLabelThread, // thread function entry point
            (void *)&(threadParams[i]) // parameters to pass in
          );
     }
     for(int i=0; i<NUM_THREADS; i++)
     {
          pthread_join(threads[i], NULL);
          countPr+=threadParams[i].countPr;
          countR+=threadParams[i].countR;
          countPh+=threadParams[i].countPh;
          countL+=threadParams[i].countL;
          countW+=threadParams[i].countW;
          if(threadParams[i].w1y < W1y)
          {
            W1x = threadParams[i].w1x;
            W1y = threadParams[i].w1y;
          }
     }

      std::string msecs = to_string(int(cap.get(CAP_PROP_POS_MSEC)));



        if(countPr > 4000)
        {
          hasProp=1;
          if(!hadProp)
          {
            outfile << "Propofol" << "," << msecs << endl;
          }
          hadProp=1;
        }
        if(countR > 4000)
        {
          hasRoc=1;
          if(!hadRoc)
          {
            outfile << "Rocuronium" << "," << msecs << endl;
          }
          hadRoc=1;
        }
        if(countPh > 8000)
        {
          hasPhen=1;
          if(!hadPhen)
          {
            outfile << "Phenylephrine" << "," << msecs << endl;
          }
          hadPhen=1;
        }
        if(countL > (hadProp*12000+hadRoc*12000+hadPhen*12000+hadDex*12000+hadOd*12000+12000))
        {
          hasLid=1;
          if(!hadLid)
          {
            outfile << "Lidocaine" << "," << msecs << endl;
          }
          hadLid=1;
        }
        if(countW > 4000)
        {
          hasOd=1;
          if(!hadOd)
          {
            outfile << "Odanestron" << "," << msecs << endl;
          }
          hadOd=1;
          hasDex=1;
          if(!hadDex)
          {
            outfile << "Dexamethasone" << "," << msecs << endl;
          }
          hadDex=1;
        }


      if(1)
      {
        if(hadProp && countPr <= 4000) 
        { 
          hadProp=0; 
          outfile << "Propofol" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
        }

        if(hadRoc && countR <= 4000) 
        { 
          hadRoc=0; 
          outfile << "Rocuronium" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
        }

        if(hadPhen && countPh <= 8000) 
        { 
          hadPhen=0; 
          outfile << "Phenylephrine" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
        }

        if(hadLid && countL < (hadProp*12000+hadRoc*12000+hadPhen*12000+hadDex*12000+hadOd*12000+12000)) 
        { 
          hadLid=0; 
          outfile << "Lidocaine" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
        }
     }
/*
        if(countW <= 2000) 
        { 
          if(hadDex)
          {
            hadDex=0; 
            outfile << "Dexamethasone" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
          }
          if(hadOd)
          {
            hadOd=0;
            outfile << "Odanestron" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
          }
        }
        else if(countW <= 4000 && ( (!hadOd && !hadDex) || (hadOd && hadDex) || (hadDex && !hadOd) ) ) 
        { 
          int ybound = (srcclone.rows-W1y > 150) ? 150 : srcclone.rows-W1y;
          dst = srcclone(Rect(W1x,W1y,300,ybound));
          pyrUp(dst,dst,Size(dst.cols*2,dst.rows*2));
          pyrUp(dst,dst,Size(dst.cols*2,dst.rows*2));
          cvtColor(dst, gray, COLOR_BGR2GRAY);
          GaussianBlur(gray, gray, Size(7, 7), 0);
          vector<Vec3f> circles;
          
          int maxthresh = threshold(gray,threshed,0,255,THRESH_BINARY + THRESH_OTSU);
          int minthresh = maxthresh / 2;
          maxthresh = maxthresh-10;
          minthresh = minthresh-5;
          maxthresh = (maxthresh < 10) ? 255: maxthresh;
          minthresh = (maxthresh == 255) ? 255: minthresh;

          Canny(gray, canned, 35, 70);

          HoughCircles(gray, circles, HOUGH_GRADIENT, 1, gray.rows/32, 70, 20, 20, 35);

          circnt+=circles.size();
          for( size_t i = 0; i < circles.size(); i++ )
          {
            Point center(cvRound(circles[i][0]), cvRound(circles[i][1]));
            int radius = cvRound(circles[i][2]);
            circle( dst, center, radius, Scalar(0,0,255), 3, 8, 0 );
            if(i < 3) 
            {
              putText(dst, to_string(radius), Point(100*(i+1),100), FONT_HERSHEY_COMPLEX_SMALL, 3, Scalar(0,0,255), 3, LINE_AA);
            }
          }

          if(circles.size() > 0)
          {
            if(hadDex)
            {
            hadDex=0; 
            outfile << "Dexamethasone" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
            }
            if(!hadOd)
            {
            hadOd=1;
            outfile << "Odanestron" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;              
            }
          }
          else
          {
            if(!hadDex)
            {
            hadDex=1; 
            outfile << "Dexamethasone" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
            }
            if(hadOd) //unnecessary should not reach
            {
            hadOd=0;
            outfile << "Odanestron" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;              
            }
          }
        }


*/
   }
 
   outfile.close();
}
