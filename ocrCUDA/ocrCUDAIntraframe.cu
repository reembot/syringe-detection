#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/imgproc.hpp"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <vector>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;
using namespace std;

#include <pthread.h>
#include <sched.h>

#define NUM_THREADS 4

typedef struct {
    int threadIdx;
    Mat roisrc;
    Mat prevroi;
    int countPr;
    int countR;
    int countPh;
    int countL;
    int countW;
    int w1x;
    int w1y;
    int framenum;
}  threadParams_t;

// POSIX thread declarations and scheduling attributes
//
pthread_t threads[NUM_THREADS];
threadParams_t threadParams[NUM_THREADS];


// See www.asciitable.com
#define ESCAPE_KEY (27)
#define SYSTEM_ERROR (-1)

__global__ void mykernel(unsigned char *roirow, unsigned char *prevroi){
    unsigned int tid = threadIdx.x;
    float bval,gval,rval,bval2,gval2,rval2;
    unsigned int i = blockIdx.x*blockDim.x+tid;
    if(i<506880 && i%3==0) 
    {
      bval=(float)roirow[i];
      gval=(float)roirow[i+1];
      rval=(float)roirow[i+2];
      bval=(float)prevroi[i];
      gval=(float)prevroi[i+1];
      rval=(float)prevroi[i+2];

//    if(abs(bval-bval2) < 30 && abs(gval-gval2) < 30 && abs(gval-gval2) < 30)
    if(1)
    {
      if ( (rval > 96) && (rval-bval > rval/3) && (bval < 32)  && (gval < rval * 9 / 10) && (gval > 3 * rval / 5) ) 
      {
        roirow[i]=1;
      }   
      else if ( (rval > 224) && (bval < 64) && (gval < 96) ) 
      {   
        roirow[i]=2; 
      } 
      else if ( (rval > 128) && (bval > 128) && (gval < 3 * rval / 4) && (gval > rval / 2) ) 
      {
        roirow[i]=3;
      } 
      else if ( (abs(rval-gval) < 20) && (abs(rval-bval) < 20) && (abs(gval-bval) < 20) ) 
      {
        if ( (rval > 48) && (gval > 48) && (bval > 48) && (rval < 80) && (gval < 80) && (bval < 80) ) 
        {
          roirow[i]=4;
        } 
        else if ( (rval > 175) && (gval > 175) && (bval > 175) ) 
        {
          roirow[i]=5;
        }
        else
        {
          roirow[i]=0;
        }
      }
      else
      {
        roirow[i]=0;
      }
    }
    else
    {
      roirow[i]=0;
    }
    }
    
}

void *findLabelThread(void *threadp)
{
    hipError_t cuda_ret;
    threadParams_t *threadParams = (threadParams_t *)threadp;
    int size = 16*sizeof(int);
    unsigned char rowvals[506880];
//    for(int i=0;i<506880;i++) rowvals[i]=0;
    unsigned char *gpuprevroi;
    unsigned char *gpuroirow;
    cuda_ret=hipMalloc((void **)&gpuroirow,506880*sizeof(unsigned char));
    cuda_ret=hipMalloc((void **)&gpuprevroi,506880*sizeof(unsigned char));
    hipMemcpy(gpuroirow,threadParams->roisrc.ptr<uchar>(threadParams->threadIdx*220),506880*sizeof(unsigned char),hipMemcpyHostToDevice);
    hipMemcpy(gpuprevroi,threadParams->prevroi.ptr<uchar>(threadParams->threadIdx*220),506880*sizeof(unsigned char),hipMemcpyHostToDevice);
//         hipDeviceSynchronize();
//    hipStreamSynchronize(0);
    mykernel<<<495,1024>>>(gpuroirow,gpuprevroi);
//    hipDeviceSynchronize();
//    hipStreamSynchronize(0);
//         cuda_ret = hipGetLastError();
    hipMemcpy(rowvals,gpuroirow,506880*sizeof(unsigned char),hipMemcpyDeviceToHost);
//    hipDeviceSynchronize();
//    hipStreamSynchronize(0);
    for(int i=0;i<506880;i+=3) 
    {
      switch(rowvals[i])
      {
        case 1:
          threadParams->countPr++;
          break;
        case 2:
          threadParams->countR++;
          break;
        case 3:
          threadParams->countPh++;
          break;
        case 4:
          threadParams->countL++;
          break;
        case 5:
          threadParams->countW++;
          break;
      }
    }

    hipFree(gpuroirow);
    hipFree(gpuprevroi);
//    hipStreamSynchronize(0);

}

int main(int argc, char** argv)
{
   const char* default_file = "0";
   const char* output_file = "vout.csv";
   const char* filename = argc >=2 ? argv[1] : default_file;
   const char* out_filename = argc >=3 ? argv[2] : output_file;
   ofstream outfile;
   outfile.open(out_filename);
   VideoCapture cap;
   int waitval = 1;
   if (*filename == '0') {
     cap.open(0);
   } else {
     cap.open(filename);
   }
   bool origView=1;
   bool hadRoc=0,hadProp=0,hadPhen=0,hadLid=0,hadOd=0,hadDex=0;
   int pW1x=0,pW1y=0,pW2x=0,pW2y=0;
   int prevStamp=0;

/*
   namedWindow(filename);
   namedWindow("zoomed");
   namedWindow("canned");
*/

   char winInput;

   Mat src, roisrc, srcSeg, srcSeg2, srcSegSum, prevroi, srcclone, dst, gray,threshed,canned,subshow;
   Mat matPr,matR,matPh,matL,matD;
   int showMat = 0;

   cap.read(src);
   if(src.empty())
   {
     return 0;
   }
   srcSeg = src.clone();
   srcSegSum = src.clone();
   dst = src.clone();
   cvtColor(dst, gray, COLOR_BGR2GRAY);
   Canny(gray, canned, 35, 70);
   int cnt=0,circnt=0;

//   VideoWriter vout;
//   Size S = Size((int) cap.get(CAP_PROP_FRAME_WIDTH),    // Acquire input size
//                  (int) cap.get(CAP_PROP_FRAME_HEIGHT));

//   vout.open("colors.mp4", VideoWriter::fourcc('H','2','6','4'), cap.get(CAP_PROP_FPS), S, true);
//   vout.open(argv[3], VideoWriter::fourcc('H','2','6','4'), 10, Size(1280,800), true);

//   printf("Press space to pause and unpause, 's' to step through each frame, 'c' to switch views or 'esc' to exit\n");

   while (1)
   {
      cnt++;
      cap.read(src);
      if(src.empty())
      {
        break;
      }
      
      srcclone = src.clone();
      roisrc = srcclone(Rect(150,75,768,880)).clone();
      matPr = roisrc.clone();
      matR = matPr.clone();
      matPh = matPr.clone();
      matL = matPr.clone();
      matD = matPr.clone();

      std::string msecs = to_string(int(cap.get(CAP_PROP_POS_MSEC)));

      float bval=0,gval=0,rval=0,bval2=0,gval2=0,rval2=0;
      
      int rowbuff = 75;
      int W1x=0,W1y=960,W2x=0,W2y=0;

      int countR=0,countPr=0,countPh=0,countL=0,countW=0;
      bool hasRoc=0,hasProp=0,hasPhen=0,hasLid=0,hasOd=0,hasDex=0;

//skip first frame to setup previous for frame differencing
      if(cnt>1) 
      {


        for(int i=0; i<NUM_THREADS; i++)
        {
          threadParams[i].threadIdx=i;
          threadParams[i].roisrc=roisrc;
          threadParams[i].prevroi=prevroi;
          threadParams[i].countPr=0;
          threadParams[i].countR=0;
          threadParams[i].countPh=0;
          threadParams[i].countL=0;
          threadParams[i].countW=0;
          threadParams[i].w1x=0;
          threadParams[i].w1y=0;
          threadParams[i].framenum=cnt;
          pthread_create(&threads[i],   // pointer to thread descriptor
//            (void *)0,     // use default attributes
            NULL,
            findLabelThread, // thread function entry point
            (void *)&(threadParams[i]) // parameters to pass in
          );
        }
        for(int i=0; i<NUM_THREADS; i++)
        {
          pthread_join(threads[i], NULL);
          countPr+=threadParams[i].countPr;
          countR+=threadParams[i].countR;
          countPh+=threadParams[i].countPh;
          countL+=threadParams[i].countL;
          countW+=threadParams[i].countW;
          if(threadParams[i].w1y < W1y)
          {
            W1x = threadParams[i].w1x;
            W1y = threadParams[i].w1y;
          }
        }
        if(countPr > 1000)
        {
          hasProp=1;
          if(!hadProp)
          {
            outfile << "Propofol" << "," << msecs << endl;
          }
          hadProp=1;
        }
        if(countR > 1000)
        {
          hasRoc=1;
          if(!hadRoc)
          {
            outfile << "Rocuronium" << "," << msecs << endl;
          }
          hadRoc=1;
        }
        if(countPh > 2000)
        {
          hasPhen=1;
          if(!hadPhen)
          {
            outfile << "Phenylephrine" << "," << msecs << endl;
          }
          hadPhen=1;
        }
        if(countL > (hadProp*3000+hadRoc*3000+hadPhen*3000+hadDex*3000+hadOd*3000+3000))
        {
          hasLid=1;
          if(!hadLid)
          {
            outfile << "Lidocaine" << "," << msecs << endl;
          }
          hadLid=1;
        }
        if(countW > 1000)
        {
          hasOd=1;
          if(!hadOd)
          {
            outfile << "Odanestron" << "," << msecs << endl;
          }
          hadOd=1;
          hasDex=1;
          if(!hadDex)
          {
            outfile << "Dexamethasone" << "," << msecs << endl;
          }
          hadDex=1;
        }

      }


//if no new medications were detected then check counts to see if one was removed
//      if (!hasProp && !hasRoc && !hasPhen && !hasLid && !hasOd && !hasDex)
      if(1)
      {
        if(hadProp && countPr <= 1000) 
        { 
          hadProp=0; 
          outfile << "Propofol" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
        }
        if(hadRoc && countR <= 1000) 
        { 
          hadRoc=0; 
          outfile << "Rocuronium" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
        }
        if(hadPhen && countPh <= 2000) 
        { 
          hadPhen=0; 
          outfile << "Phenylephrine" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
        }
        if(hadLid && countL < (hadProp*3000+hadRoc*3000+hadPhen*3000+hadDex*3000+hadOd*3000+3000)) 
        { 
          hadLid=0; 
          outfile << "Lidocaine" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
        }
        if(countW <= 500) 
        { 
          if(hadDex)
          {
            hadDex=0; 
            outfile << "Dexamethasone" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
          }
          if(hadOd)
          {
            hadOd=0;
            outfile << "Odanestron" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
          }
        }
        else if(countW <= 1000 && ( (!hadOd && !hadDex) || (hadOd && hadDex) || (hadDex && !hadOd) ) ) 
        { 
          int ybound = (srcclone.rows-W1y > 150) ? 150 : srcclone.rows-W1y;
          dst = srcclone(Rect(W1x,W1y,300,ybound));
          pyrUp(dst,dst,Size(dst.cols*2,dst.rows*2));
          pyrUp(dst,dst,Size(dst.cols*2,dst.rows*2));
          cvtColor(dst, gray, COLOR_BGR2GRAY);
          GaussianBlur(gray, gray, Size(7, 7), 0);
          vector<Vec3f> circles;
          
          int maxthresh = threshold(gray,threshed,0,255,THRESH_BINARY + THRESH_OTSU);
          int minthresh = maxthresh / 2;
          maxthresh = maxthresh-10;
          minthresh = minthresh-5;
          maxthresh = (maxthresh < 10) ? 255: maxthresh;
          minthresh = (maxthresh == 255) ? 255: minthresh;
  //      Canny(gray, canned, minthresh, maxthresh+20);

          Canny(gray, canned, 35, 70);

          HoughCircles(gray, circles, HOUGH_GRADIENT, 1, gray.rows/32, 70, 20, 20, 35);

//          HoughCircles(gray, circles, HOUGH_GRADIENT, 1, gray.rows/32, 60, 20, 20, 30); //60D,80O
//          HoughCircles(gray, circles, HOUGH_GRADIENT, 1, gray.rows/32, 50, 20, 20, 30);
//          HoughCircles(gray, circles, HOUGH_GRADIENT, 1, gray.rows/32, maxthresh, 20, 20, 30);

          circnt+=circles.size();
          for( size_t i = 0; i < circles.size(); i++ )
          {
            Point center(cvRound(circles[i][0]), cvRound(circles[i][1]));
            int radius = cvRound(circles[i][2]);
            circle( dst, center, radius, Scalar(0,0,255), 3, 8, 0 );
            if(i < 3) 
            {
              putText(dst, to_string(radius), Point(100*(i+1),100), FONT_HERSHEY_COMPLEX_SMALL, 3, Scalar(0,0,255), 3, LINE_AA);
            }
          }

//       resize(dst,subshow,Size(600,300));
//       imshow("zoomed",subshow);
//       resize(canned,subshow,Size(600,300));
//       imshow("canned",subshow);

          if(circles.size() > 0)
          {
            if(hadDex)
            {
            hadDex=0; 
            outfile << "Dexamethasone" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
            }
            if(!hadOd)
            {
            hadOd=1;
            outfile << "Odanestron" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;              
            }
          }
          else
          {
            if(!hadDex)
            {
            hadDex=1; 
            outfile << "Dexamethasone" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;
            }
            if(hadOd) //unnecessary should not reach
            {
            hadOd=0;
            outfile << "Odanestron" << "," << cap.get(CAP_PROP_POS_MSEC) << endl;              
            }
          }
        }
      }


      if(hadRoc) {
        putText(src, "Rocuronium", Point(30,100), FONT_HERSHEY_COMPLEX_SMALL, 0.8, Scalar(0,0,255), 1, LINE_AA);
      }
      if(hadProp) {
        putText(src, "Propofol", Point(30,50), FONT_HERSHEY_COMPLEX_SMALL, 0.8, Scalar(0,0,255), 1, LINE_AA);
      }
      if(hadPhen) {
        putText(src, "Phenylephrine", Point(30,150), FONT_HERSHEY_COMPLEX_SMALL, 0.8, Scalar(0,0,255), 1, LINE_AA);
      }

      if(hadLid) { 
        putText(src, "Lidocaine", Point(30,200), FONT_HERSHEY_COMPLEX_SMALL, 0.8, Scalar(0,0,255), 1, LINE_AA); 
      }

      if(hadOd) { 
        putText(src, "Odanestron", Point(30,250), FONT_HERSHEY_COMPLEX_SMALL, 0.8, Scalar(0,0,255), 1, LINE_AA); 
      }

      if(hadDex) { 
        putText(src, "Dexamethasone", Point(30,300), FONT_HERSHEY_COMPLEX_SMALL, 0.8, Scalar(0,0,255), 1, LINE_AA); 
      }

//      showMat = 0;
//      if(showMat == 0) { srcSegSum = matPr.clone(); }
/*
      if(showMat == 1) { srcSegSum = matR.clone(); }
      if(showMat == 2) { srcSegSum = matPh.clone(); }
      if(showMat == 3) { srcSegSum = matL.clone(); }
      if(showMat == 4) { srcSegSum = matD.clone(); }
*/
//      srcSegSum = dst.clone();

      
/*
      if (origView)
      {
        imshow(filename,src);
      } else {
        imshow(filename,srcSegSum);
      }
*/
      
/*
      Mat colorcanned,resizedsrc, resizedseg, expanddst, expandcanned; 
      Mat combinedh1,combinedh2,combinedv;
      cvtColor(canned, colorcanned, COLOR_GRAY2BGR);
      resize(dst,expanddst,Size(640,320));
      resize(colorcanned,expandcanned,Size(640,320));
      resize(src,resizedsrc,Size(640,480));
      resize(srcSegSum,resizedseg,Size(640,480));
      hconcat(resizedsrc,resizedseg,combinedh1);
      hconcat(expanddst,expandcanned,combinedh2);
      vconcat(combinedh1,combinedh2,combinedv);

//      imshow(filename,combinedv);
//      imshow(filename,src);
      vout.write(combinedv);
*/
      prevroi = roisrc.clone();

//      vout.write(src);

/*
      if ((winInput = waitKey(waitval)) == ESCAPE_KEY)
      {
          break;
      }
      else if(winInput == 's') 
      {
          waitval=0;
      }
      else if(winInput == ' ') 
      {
          waitval = (waitval == 100) ? 0 : 100;
      }
      else if(winInput == 'c') 
      {
          origView = origView ? 0 : 1; //flip flag to show original or difference
      }
      else if(winInput == 'm') 
      {
          showMat = (showMat > 3) ? 0 : showMat + 1; //flip flag to show original or difference
      }
      else if(winInput == 'p') 
      {
          String imgname = filename + to_string(cnt) + ".png";
          imwrite(imgname, src);
      }
*/

   }
//   destroyWindow(filename); 
   outfile.close();
}
